#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

// NVTX workaround for Windows environments
#ifdef _WIN32
    #define NVTX_DISABLE  // Temporarily disable NVTX on Windows
#endif

#ifdef NVTX_DISABLE
    // Define dummy NVTX macros for Windows platforms
    #define nvtxRangePush(name) 
    #define nvtxRangePop()
#else
    #include <nvtx3/nvToolsExt.h>
#endif

namespace cg = cooperative_groups;

// Kernel for efficient batched GEMV operations with small vector dimensions (16–128)
__global__ void batched_gemv_kernel(
    const float* __restrict__ weights,    // Shape: [batch_size, input_dim, output_dim]
    const float* __restrict__ inputs,     // Shape: [batch_size, input_dim]
    float* __restrict__ outputs,          // Shape: [batch_size, output_dim]
    int batch_size,
    int input_dim,
    int output_dim
) {
    int batch_idx = blockIdx.x;
    int output_idx = threadIdx.x;
    
    if (batch_idx >= batch_size || output_idx >= output_dim) return;
    
    // Shared memory allocation for input vectors to optimize memory access
    extern __shared__ float shared_input[];
    
    auto block = cg::this_thread_block();
    
    // Load input vector elements into shared memory for coalesced access
    if (output_idx < input_dim) {
        shared_input[output_idx] = inputs[batch_idx * input_dim + output_idx];
    }
    block.sync();
    
    // Compute the dot product for the current output dimension
    float result = 0.0f;
    const float* weight_row = weights + (batch_idx * input_dim + 0) * output_dim + output_idx;
    
    #pragma unroll 8
    for (int i = 0; i < input_dim; i++) {
        result += shared_input[i] * weight_row[i * output_dim];
    }
    
    outputs[batch_idx * output_dim + output_idx] = result;
}

// Softmax kernel with optimized single-pass reduction
__global__ void batched_softmax_kernel(
    const float* __restrict__ inputs,     // Shape: [batch_size, dim]
    float* __restrict__ outputs,          // Shape: [batch_size, dim]
    int batch_size,
    int dim
) {
    int batch_idx = blockIdx.x;
    int tid = threadIdx.x;
    
    if (batch_idx >= batch_size) return;
    
    extern __shared__ float sdata[];
    float* shared_max = sdata;
    float* shared_sum = sdata + blockDim.x;
    
    const float* input_batch = inputs + batch_idx * dim;
    float* output_batch = outputs + batch_idx * dim;
    
    auto block = cg::this_thread_block();
    
    // Parallel reduction to determine the maximum value for numerical stability
    float local_max = -INFINITY;
    for (int i = tid; i < dim; i += blockDim.x) {
        local_max = fmaxf(local_max, input_batch[i]);
    }
    shared_max[tid] = local_max;
    block.sync();
    
    // Reduce to obtain the global maximum value
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_max[tid] = fmaxf(shared_max[tid], shared_max[tid + stride]);
        }
        block.sync();
    }
    float global_max = shared_max[0];
    
    // Calculate exponentials and their sum for normalization
    float local_sum = 0.0f;
    for (int i = tid; i < dim; i += blockDim.x) {
        float exp_val = expf(input_batch[i] - global_max);
        output_batch[i] = exp_val;
        local_sum += exp_val;
    }
    shared_sum[tid] = local_sum;
    block.sync();
    
    // Reduce to compute the total sum of exponentials
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_sum[tid] += shared_sum[tid + stride];
        }
        block.sync();
    }
    float total_sum = shared_sum[0];
    
    // Normalize outputs to obtain softmax probabilities
    for (int i = tid; i < dim; i += blockDim.x) {
        output_batch[i] /= total_sum;
    }
}

// Kernel for high-throughput processing of price vectors
__global__ void process_price_vectors_kernel(
    const float* __restrict__ prices,     // Shape: [batch_size, n_assets]
    const float* __restrict__ weights,    // Shape: [n_assets, n_features]
    float* __restrict__ features,         // Shape: [batch_size, n_features]
    int batch_size,
    int n_assets,
    int n_features
) {
    int batch_idx = blockIdx.x;
    int feature_idx = threadIdx.x;
    
    if (batch_idx >= batch_size || feature_idx >= n_features) return;
    
    const float* price_vector = prices + batch_idx * n_assets;
    float result = 0.0f;
    
    // Vectorized dot product computation with manual unrolling for efficiency
    #pragma unroll 4
    for (int i = 0; i < n_assets; i++) {
        result += price_vector[i] * weights[i * n_features + feature_idx];
    }
    
    features[batch_idx * n_features + feature_idx] = result;
}

// C interface for integration with Python environments
extern "C" {

void launch_batched_gemv(
    const float* weights, const float* inputs, float* outputs,
    int batch_size, int input_dim, int output_dim,
    hipStream_t stream = 0
) {
    nvtxRangePush("batched_gemv");
    
    dim3 grid(batch_size);
    dim3 block(min(output_dim, 1024));
    int shared_mem = input_dim * sizeof(float);
    
    batched_gemv_kernel<<<grid, block, shared_mem, stream>>>(
        weights, inputs, outputs, batch_size, input_dim, output_dim
    );
    
    nvtxRangePop();
}

void launch_batched_softmax(
    const float* inputs, float* outputs,
    int batch_size, int dim,
    hipStream_t stream = 0
) {
    nvtxRangePush("batched_softmax");
    
    dim3 grid(batch_size);
    dim3 block(min(dim, 1024));
    int shared_mem = 2 * block.x * sizeof(float);
    
    batched_softmax_kernel<<<grid, block, shared_mem, stream>>>(
        inputs, outputs, batch_size, dim
    );
    
    nvtxRangePop();
}

void launch_price_vector_processing(
    const float* prices, const float* weights, float* features,
    int batch_size, int n_assets, int n_features,
    hipStream_t stream = 0
) {
    nvtxRangePush("price_vectors");
    
    dim3 grid(batch_size);
    dim3 block(min(n_features, 1024));
    
    process_price_vectors_kernel<<<grid, block, 0, stream>>>(
        prices, weights, features, batch_size, n_assets, n_features
    );
    
    nvtxRangePop();
}

} // extern "C"
